#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <vector>

constexpr uint32_t P = 2147483647; // 2 ** 31 - 1

// CUDA Kernel for hierarchical folding
__global__ void foldKernel(uint64_t* values, const uint64_t* factors, int n, int numFactors, int level)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n) {
    // Step size doubles at each level
    int step = 1 << (level + 1);
    int invlevel = numFactors - level - 1;

    if (idx < n / step) {
      // Compute indices for the pair to be reduced
      int leftIdx = idx * step;
      int rightIdx = leftIdx + step / 2;

      // Perform the folding operation
      values[leftIdx] = (values[leftIdx] + (factors[invlevel] * values[rightIdx])) % P;
    }
  }
}

template <typename E, typename S>
__global__ void
fold_kernel_2(S* values, const E* folding_factors, const int level, const int nlog2, const int n, E* result)
{
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
  int step = 1 << (nlog2 - level - 1);
  int offset = idx + step;
  if (offset + (step - 1) < n) {
    result[idx] = (values[idx] + (folding_factors[level] * values[offset])) % P;
  }
  __syncthreads();
}

// Wrapper function to handle CUDA kernel invocation
void hierarchicalFold(std::vector<uint64_t>& values, const std::vector<uint64_t>& factors)
{
  int n = values.size();
  int numFactors = factors.size();

  // Validate input
  if ((1 << numFactors) != n) {
    throw std::invalid_argument("Number of elements must be a power of 2 matching the number of factors.");
  }

  // Allocate device memory
  uint64_t* d_values;
  uint64_t* d_factors;
  hipMalloc(&d_values, n * sizeof(uint64_t));
  hipMalloc(&d_factors, numFactors * sizeof(uint64_t));

  // Copy data to device
  hipMemcpy(d_values, values.data(), n * sizeof(uint64_t), hipMemcpyHostToDevice);
  hipMemcpy(d_factors, factors.data(), numFactors * sizeof(uint64_t), hipMemcpyHostToDevice);

  int blockSize = 1024;

  int nlog2 = log2(n);

  // Launch kernel
  int gridSize = (n + blockSize - 1) / blockSize;

  auto start = std::chrono::high_resolution_clock::now();
  for (int level = 0; level < numFactors; ++level) {
    foldKernel<<<gridSize, blockSize>>>(d_values, d_factors, n, numFactors, level);
  }
  auto end = std::chrono::high_resolution_clock::now();

  std::chrono::duration<double, std::milli> elapsed = end - start;
  std::cout << "time for 2^" << numFactors << " is: " << elapsed.count() << " ms" << std::endl;
  // Copy result back to host
  hipMemcpy(values.data(), d_values, n * sizeof(uint64_t), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_values);
  hipFree(d_factors);
}

int main()
{
  // Example input
  // std::vector<uint64_t> values = {1, 2, 3, 4, 5, 6, 7, 8};
  // std::vector<uint64_t> factors = {2, 3, 4}; // Folding factors

  // Set the length of factors
  const size_t factors_length = 20;                 // Example length
  const size_t values_length = 1 << factors_length; // 2^factors_length

  // Initialize the `values` vector
  std::vector<uint64_t> values(values_length);
  for (size_t i = 0; i < values_length; ++i) {
    values[i] = static_cast<uint64_t>(i + 1); // Populate with 1, 2, ..., values_length
  }

  // Initialize the `factors` vector
  std::vector<uint64_t> factors(factors_length);
  for (size_t i = 0; i < factors_length; ++i) {
    factors[i] = static_cast<uint64_t>(i + 2); // Populate with 2, 3, ..., factors_length + 1
  }

  try {
    for (size_t i = 0; i < 1; ++i) {
      for (size_t j = 0; j < values_length; ++j) {
        values[j] = static_cast<uint64_t>(j + 1); // Populate with 1, 2, ..., values_length
      }
      hierarchicalFold(values, factors);
      // if (values[0] != 65782334) {
      //   std::cout << "Error: " << values[0] << " " << i << std::endl;

      //   return;
      // }
    }
    std::cout << "Folded result: ";
    for (uint64_t v : values) {
      std::cout << v << " ";
      if (values_length > 128) break;
    }
    std::cout << std::endl;
  } catch (const std::exception& e) {
    std::cerr << "Error: " << e.what() << std::endl;
  }

  return 0;
}
