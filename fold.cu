#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

constexpr uint32_t P = 2147483647; // 2 ** 31 - 1

// CUDA Kernel for hierarchical folding
__global__ void foldKernel(uint64_t* values, const uint64_t* factors, int n, int numFactors)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n) {
    for (int level = 0; level < numFactors; ++level) {
      // Step size doubles at each level
      int step = 1 << (level + 1);
      int invlevel = numFactors - level - 1;

      if (idx < n / step) {
        // Compute indices for the pair to be reduced
        int leftIdx = idx * step;
        int rightIdx = leftIdx + step / 2;

        // printf("Folding %f and %f * %f\n", values[leftIdx], factors[invlevel], values[rightIdx]);

        // Perform the folding operation
        values[leftIdx] = (values[leftIdx] + (factors[invlevel] * values[rightIdx])) % P;
        // values[leftIdx] = (values[leftIdx] + (factors[invlevel]  * values[rightIdx])% P) % P;
      }

      // Synchronize threads to ensure all reductions for the current level are complete
      __syncthreads();
    }
  }
}

// Wrapper function to handle CUDA kernel invocation
void hierarchicalFold(std::vector<uint64_t>& values, const std::vector<uint64_t>& factors)
{
  int n = values.size();
  int numFactors = factors.size();

  // Validate input
  if ((1 << numFactors) != n) {
    throw std::invalid_argument("Number of elements must be a power of 2 matching the number of factors.");
  }

  // Allocate device memory
  uint64_t* d_values;
  uint64_t* d_factors;
  hipMalloc(&d_values, n * sizeof(uint64_t));
  hipMalloc(&d_factors, numFactors * sizeof(uint64_t));

  // Copy data to device
  hipMemcpy(d_values, values.data(), n * sizeof(uint64_t), hipMemcpyHostToDevice);
  hipMemcpy(d_factors, factors.data(), numFactors * sizeof(uint64_t), hipMemcpyHostToDevice);

  // Launch kernel
  int blockSize = 1024;

  int gridSize = (n + blockSize - 1) / blockSize;

  foldKernel<<<gridSize, blockSize>>>(d_values, d_factors, n, numFactors);

  // Copy result back to host
  hipMemcpy(values.data(), d_values, n * sizeof(uint64_t), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_values);
  hipFree(d_factors);
}

int main()
{
  // Example input
  // std::vector<uint64_t> values = {1, 2, 3, 4, 5, 6, 7, 8};
  // std::vector<uint64_t> factors = {2, 3, 4}; // Folding factors

  // Set the length of factors
  const size_t factors_length = 12;                  // Example length
  const size_t values_length = 1 << factors_length; // 2^factors_length

  // Initialize the `values` vector
  std::vector<uint64_t> values(values_length);
  for (size_t i = 0; i < values_length; ++i) {
    values[i] = static_cast<uint64_t>(i + 1); // Populate with 1, 2, ..., values_length
  }

  // Initialize the `factors` vector
  std::vector<uint64_t> factors(factors_length);
  for (size_t i = 0; i < factors_length; ++i) {
    factors[i] = static_cast<uint64_t>(i + 2); // Populate with 2, 3, ..., factors_length + 1
  }

  try {
    for (size_t i = 0; i < 1000; ++i) {
      for (size_t j = 0; j < values_length; ++j) {
        values[j] = static_cast<uint64_t>(j + 1); // Populate with 1, 2, ..., values_length
      }
      hierarchicalFold(values, factors);
      if (values[0] != 65782334) {
        std::cout << "Error: " << values[0] << " " << i << std::endl;

        return;
      }
    }
    std::cout << "Folded result: ";
    for (uint64_t v : values) {
      std::cout << v << " ";
      if (values_length > 128) break;
    }
    std::cout << std::endl;
  } catch (const std::exception& e) {
    std::cerr << "Error: " << e.what() << std::endl;
  }

  return 0;
}
