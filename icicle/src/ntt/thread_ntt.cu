#include "hip/hip_runtime.h"
#ifndef T_NTT
#define T_NTT
#pragma once

#include <stdio.h>
#include <stdint.h>
#include "gpu-utils/modifiers.cuh"

struct stage_metadata {
  uint32_t th_stride;
  uint32_t ntt_block_size;
  uint32_t batch_id;
  uint32_t ntt_block_id;
  uint32_t ntt_inp_id;
};

#define STAGE_SIZES_DATA                                                                                               \
  {                                                                                                                    \
    {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {4, 0, 0, 0, 0}, {5, 0, 0, 0, 0},              \
      {6, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {4, 4, 0, 0, 0}, {5, 4, 0, 0, 0}, {5, 5, 0, 0, 0}, {6, 5, 0, 0, 0},            \
      {6, 6, 0, 0, 0}, {4, 5, 4, 0, 0}, {4, 6, 4, 0, 0}, {5, 5, 5, 0, 0}, {6, 4, 6, 0, 0}, {6, 5, 6, 0, 0},            \
      {6, 6, 6, 0, 0}, {6, 5, 4, 4, 0}, {5, 5, 5, 5, 0}, {6, 5, 5, 5, 0}, {6, 5, 5, 6, 0}, {6, 6, 6, 5, 0},            \
      {6, 6, 6, 6, 0}, {5, 5, 5, 5, 5}, {6, 5, 4, 5, 6}, {6, 5, 5, 5, 6}, {6, 5, 6, 5, 6}, {6, 6, 5, 6, 6},            \
      {6, 6, 6, 6, 6},                                                                                                 \
  }
uint32_t constexpr STAGE_SIZES_HOST[31][5] = STAGE_SIZES_DATA;
__device__ constexpr uint32_t STAGE_SIZES_DEVICE[31][5] = STAGE_SIZES_DATA;

// construction for fast-twiddles
uint32_t constexpr STAGE_PREV_SIZES[31] = {0,  0,  0,  0,  0,  0,  0,  0,  4,  5,  5,  6,  6,  9,  9, 10,
                                           11, 11, 12, 15, 15, 16, 16, 18, 18, 20, 21, 21, 22, 23, 24};

#define STAGE_SIZES_DATA_FAST_TW                                                                                       \
  {                                                                                                                    \
    {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {4, 0, 0, 0, 0}, {5, 0, 0, 0, 0},              \
      {6, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {4, 4, 0, 0, 0}, {5, 4, 0, 0, 0}, {5, 5, 0, 0, 0}, {6, 5, 0, 0, 0},            \
      {6, 6, 0, 0, 0}, {5, 4, 4, 0, 0}, {5, 4, 5, 0, 0}, {5, 5, 5, 0, 0}, {6, 5, 5, 0, 0}, {6, 5, 6, 0, 0},            \
      {6, 6, 6, 0, 0}, {5, 5, 5, 4, 0}, {5, 5, 5, 5, 0}, {6, 5, 5, 5, 0}, {6, 5, 5, 6, 0}, {6, 6, 6, 5, 0},            \
      {6, 6, 6, 6, 0}, {5, 5, 5, 5, 5}, {6, 5, 5, 5, 5}, {6, 5, 5, 5, 6}, {6, 5, 5, 6, 6}, {6, 6, 6, 5, 6},            \
      {6, 6, 6, 6, 6},                                                                                                 \
  }
uint32_t constexpr STAGE_SIZES_HOST_FT[31][5] = STAGE_SIZES_DATA_FAST_TW;
__device__ uint32_t constexpr STAGE_SIZES_DEVICE_FT[31][5] = STAGE_SIZES_DATA_FAST_TW;

template <typename E, typename S>
class DCCTEngine
{
public:
  E X[8];
  S WB[12];

  DEVICE_INLINE void loadBasicTwiddlesGeneric(
    S* basic_twiddles,
    uint32_t tw_order,
    uint32_t tw_log_order,
    stage_metadata s_meta,
    uint32_t tw_log_size,
    uint32_t twiddles_offset,
    uint32_t ntt_log_size,
    bool inv,
    bool dit,
    bool phase)
  {
    size_t stage_size = 1 << (tw_log_size - 1);
    size_t tw_size = (1 << tw_log_size) * tw_log_size;
    uint32_t blocks_per_batch = (1 << (tw_log_size - ntt_log_size)) - 1;
    uint32_t phase_offset = stage_size * phase * (dit ? ntt_log_size - 3 : 3); // 3 is the number of stages
    uint32_t block_offset;
    if (tw_log_order) {
      block_offset = (s_meta.ntt_block_id & (tw_order - 1)) +
                     ((s_meta.ntt_block_id & blocks_per_batch) >> tw_log_order) *
                     (1 << tw_log_order - 1) *
                     s_meta.ntt_block_size;
    } else {
      block_offset = (s_meta.ntt_block_id & blocks_per_batch) * (1 << ntt_log_size - 1);
    }
    uint32_t ntt_inp_offset = s_meta.ntt_inp_id * (1 << tw_log_order) * ((phase != dit) ? 4 : 1);
    uint32_t base_exp = phase_offset + twiddles_offset + block_offset + ntt_inp_offset;
    uint32_t exp;

    UNROLL
    for (int stage = 0; stage < ((phase != dit) ? ntt_log_size - 3 : 3); stage++) {
      UNROLL
      for (int i = 0; i < 4; i++) {
        exp = base_exp + i * (tw_order ? tw_order : 1) * (1 << ((phase != dit) ? 0 : ntt_log_size - 3));

        WB[stage * 4 + i] = basic_twiddles[inv ? (tw_size - 1 - exp) : exp];
      }
      base_exp += stage_size;
    }
  }

  DEVICE_INLINE void
  loadGlobalData(const E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id;
    }

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      X[i] = data[s_meta.th_stride * i * data_stride_u64];
    }
  }
  DEVICE_INLINE void
  loadGlobalData64(const E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id * 8 +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id * 8;
    }

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      X[i] = data[i * data_stride_u64];
    }
  }

  DEVICE_INLINE void
  storeGlobalData(E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id * 8 +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id * 8;
    }

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      data[i * data_stride_u64] = X[i];
    }
  }

  DEVICE_INLINE void
  storeGlobalDataDit(E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id;
    }

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      data[i * 8 * data_stride_u64] = X[i];
    }
  }

  DEVICE_INLINE void
  loadGlobalData32(const E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id * 8 +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id * 8;
    }

    UNROLL
    for (uint32_t j = 0; j < 2; j++) {
      UNROLL
      for (uint32_t i = 0; i < 4; i++) {
        X[4 * j + i] = data[(4 * j + i) * data_stride_u64];
      }
    }
  }

  DEVICE_INLINE void
  storeGlobalData32dit(E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id;
    }

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      data[i * 4 * data_stride_u64] = X[i];
    }
  }

  DEVICE_INLINE void
  storeGlobalData32(E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id * 8 +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id * 8;
    }

    UNROLL
    for (uint32_t j = 0; j < 2; j++) {
      UNROLL
      for (uint32_t i = 0; i < 4; i++) {
        data[(4 * j + i) * data_stride_u64] = X[4 * j + i];
      }
    }
  }

  DEVICE_INLINE void
  loadGlobalData16(const E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id * 8 +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id * 8;
    }

    UNROLL
    for (uint32_t j = 0; j < 4; j++) {
      UNROLL
      for (uint32_t i = 0; i < 2; i++) {
        X[2 * j + i] = data[(2 * j + i) * data_stride_u64];
      }
    }
  }

  DEVICE_INLINE void
  storeGlobalData16(E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id * 8 +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id * 8;
    }

    UNROLL
    for (uint32_t j = 0; j < 4; j++) {
      UNROLL
      for (uint32_t i = 0; i < 2; i++) {
        data[(2 * j + i) * data_stride_u64] = X[2 * j + i];
      }
    }
  }

  DEVICE_INLINE void
  storeGlobalData16dit(E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id;
    }

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      data[i * 2 * data_stride_u64] = X[i];
    }
  }

#define BF(t, x, y)                                                                                                    \
  t = x;                                                                                                               \
  x = x + y;                                                                                                           \
  y = t - y;

  DEVICE_INLINE void ntt4_2()
  {
    E T;

    // Stage 0
    X[2] = X[2] * WB[0];
    X[3] = X[3] * WB[1];
    BF(T, X[0], X[2]);
    BF(T, X[1], X[3]);

    X[6] = X[6] * WB[2];
    X[7] = X[7] * WB[3];
    BF(T, X[4], X[6]);
    BF(T, X[5], X[7]);

    // Stage 1
    X[1] = X[1] * WB[4];
    X[3] = X[3] * WB[5];
    BF(T, X[0], X[1]);
    BF(T, X[2], X[3]);

    X[5] = X[5] * WB[6];
    X[7] = X[7] * WB[7];
    BF(T, X[4], X[5]);
    BF(T, X[6], X[7]);
  }

  DEVICE_INLINE void ntt2_4()
  {
    E T;

    UNROLL
    for (int i = 0; i < 4; i++) {
      X[2 * i + 1] = X[2 * i + 1] * WB[i];
      BF(T, X[2 * i], X[2 * i + 1]);
    }
  }

  DEVICE_INLINE void ntt8()
  {
    E T;

    // Stage 0
    X[4] = X[4] * WB[0];
    X[5] = X[5] * WB[1];
    X[6] = X[6] * WB[2];
    X[7] = X[7] * WB[3];

    BF(T, X[0], X[4]);
    BF(T, X[1], X[5]);
    BF(T, X[2], X[6]);
    BF(T, X[3], X[7]);

    // Stage 1
    X[2] = X[2] * WB[4];
    X[3] = X[3] * WB[5];
    X[6] = X[6] * WB[6];
    X[7] = X[7] * WB[7];

    BF(T, X[0], X[2]);
    BF(T, X[1], X[3]);
    BF(T, X[4], X[6]);
    BF(T, X[5], X[7]);

    // Stage 2
    X[1] = X[1] * WB[8];
    X[3] = X[3] * WB[9];
    X[5] = X[5] * WB[10];
    X[7] = X[7] * WB[11];

    BF(T, X[0], X[1]);
    BF(T, X[2], X[3]);
    BF(T, X[4], X[5]);
    BF(T, X[6], X[7]);
  }

#define IBF(t, x, y, tw)                                                                                               \
  t = x;                                                                                                               \
  x = x + y;                                                                                                           \
  y = (t - y) * tw;

  DEVICE_INLINE void intt4_2()
  {
    E T;

    // Stage 0
    IBF(T, X[0], X[1], WB[0]);
    IBF(T, X[2], X[3], WB[1]);

    IBF(T, X[4], X[5], WB[2]);
    IBF(T, X[6], X[7], WB[3]);

    // Stage 1
    IBF(T, X[0], X[2], WB[4]);
    IBF(T, X[1], X[3], WB[5]);

    IBF(T, X[4], X[6], WB[6]);
    IBF(T, X[5], X[7], WB[7]);
  }

  DEVICE_INLINE void intt2_4()
  {
    E T;

    UNROLL
    for (int i = 0; i < 4; i++) {
      IBF(T, X[2 * i], X[2 * i + 1], WB[i]);
    }
  }

  DEVICE_INLINE void intt8()
  {
    E T;

    // Stage 0
    IBF(T, X[0], X[1], WB[0]);
    IBF(T, X[2], X[3], WB[1]);
    IBF(T, X[4], X[5], WB[2]);
    IBF(T, X[6], X[7], WB[3]);

    // Stage 1
    IBF(T, X[0], X[2], WB[4]);
    IBF(T, X[1], X[3], WB[5]);
    IBF(T, X[4], X[6], WB[6]);
    IBF(T, X[5], X[7], WB[7]);

    // Stage 2
    IBF(T, X[0], X[4], WB[8]);
    IBF(T, X[1], X[5], WB[9]);
    IBF(T, X[2], X[6], WB[10]);
    IBF(T, X[3], X[7], WB[11]);
  }

  DEVICE_INLINE void SharedData64Columns8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x7 : threadIdx.x >> 3;
    uint32_t column_id = stride ? threadIdx.x >> 3 : threadIdx.x & 0x7;

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 64 + i * 8 + column_id] = X[i];
      } else {
        X[i] = shmem[ntt_id * 64 + i * 8 + column_id];
      }
    }
  }

  DEVICE_INLINE void SharedData64Rows8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x7 : threadIdx.x >> 3;
    uint32_t row_id = stride ? threadIdx.x >> 3 : threadIdx.x & 0x7;

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 64 + row_id * 8 + i] = X[i];
      } else {
        X[i] = shmem[ntt_id * 64 + row_id * 8 + i];
      }
    }
  }

  DEVICE_INLINE void SharedData32Columns8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0xf : threadIdx.x >> 2;
    uint32_t column_id = stride ? threadIdx.x >> 4 : threadIdx.x & 0x3;

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 32 + i * 4 + column_id] = X[i];
      } else {
        X[i] = shmem[ntt_id * 32 + i * 4 + column_id];
      }
    }
  }

  DEVICE_INLINE void SharedData32Rows8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0xf : threadIdx.x >> 2;
    uint32_t row_id = stride ? threadIdx.x >> 4 : threadIdx.x & 0x3;

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 32 + row_id * 8 + i] = X[i];
      } else {
        X[i] = shmem[ntt_id * 32 + row_id * 8 + i];
      }
    }
  }

  DEVICE_INLINE void SharedData32Columns4_2(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0xf : threadIdx.x >> 2;
    uint32_t column_id = (stride ? threadIdx.x >> 4 : threadIdx.x & 0x3) * 2;

    UNROLL
    for (uint32_t j = 0; j < 2; j++) {
      UNROLL
      for (uint32_t i = 0; i < 4; i++) {
        if (store) {
          shmem[ntt_id * 32 + i * 8 + column_id + j] = X[4 * j + i];
        } else {
          X[4 * j + i] = shmem[ntt_id * 32 + i * 8 + column_id + j];
        }
      }
    }
  }

  DEVICE_INLINE void SharedData32Rows4_2(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0xf : threadIdx.x >> 2;
    uint32_t row_id = (stride ? threadIdx.x >> 4 : threadIdx.x & 0x3) * 2;

    UNROLL
    for (uint32_t j = 0; j < 2; j++) {
      UNROLL
      for (uint32_t i = 0; i < 4; i++) {
        if (store) {
          shmem[ntt_id * 32 + row_id * 4 + 4 * j + i] = X[4 * j + i];
        } else {
          X[4 * j + i] = shmem[ntt_id * 32 + row_id * 4 + 4 * j + i];
        }
      }
    }
  }

  DEVICE_INLINE void SharedData16Columns8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x1f : threadIdx.x >> 1;
    uint32_t column_id = stride ? threadIdx.x >> 5 : threadIdx.x & 0x1;

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 16 + i * 2 + column_id] = X[i];
      } else {
        X[i] = shmem[ntt_id * 16 + i * 2 + column_id];
      }
    }
  }

  DEVICE_INLINE void SharedData16Rows8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x1f : threadIdx.x >> 1;
    uint32_t row_id = stride ? threadIdx.x >> 5 : threadIdx.x & 0x1;

    UNROLL
    for (uint32_t j = 0; j < 4; j++) {
      UNROLL
      for (uint32_t i = 0; i < 2; i++) {
        if (store) {
          shmem[ntt_id * 16 + row_id * 8 + i] = X[i];
        } else {
          X[j * 2 + i] = shmem[ntt_id * 16 + row_id * 8 + j * 2 + i];
        }
      }
    }
  }

  DEVICE_INLINE void SharedData16Columns2_4(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x1f : threadIdx.x >> 1;
    uint32_t column_id = (stride ? threadIdx.x >> 5 : threadIdx.x & 0x1) * 4;

    UNROLL
    for (uint32_t j = 0; j < 4; j++) {
      UNROLL
      for (uint32_t i = 0; i < 2; i++) {
        if (store) {
          shmem[ntt_id * 16 + i * 8 + column_id + j] = X[2 * j + i];
        } else {
          X[2 * j + i] = shmem[ntt_id * 16 + i * 8 + column_id + j];
        }
      }
    }
  }

  DEVICE_INLINE void SharedData16Rows2_4(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x1f : threadIdx.x >> 1;
    uint32_t row_id = (stride ? threadIdx.x >> 5 : threadIdx.x & 0x1) * 4;

    UNROLL
    for (uint32_t j = 0; j < 4; j++) {
      UNROLL
      for (uint32_t i = 0; i < 2; i++) {
        if (store) {
          shmem[ntt_id * 16 + row_id * 2 + 2 * j + i] = X[2 * j + i];
        } else {
          X[2 * j + i] = shmem[ntt_id * 16 + row_id * 2 + 2 * j + i];
        }
      }
    }
  }
};

template <typename E, typename S>
class NTTEngine
{
public:
  E X[8];
  S WB[3];
  S WI[7];
  S WE[8];

  DEVICE_INLINE void loadBasicTwiddles(S* basic_twiddles)
  {
    UNROLL
    for (int i = 0; i < 3; i++) {
      WB[i] = basic_twiddles[i];
    }
  }

  DEVICE_INLINE void loadBasicTwiddlesGeneric(S* basic_twiddles, bool inv)
  {
    UNROLL
    for (int i = 0; i < 3; i++) {
      WB[i] = basic_twiddles[inv ? i + 3 : i];
    }
  }

  DEVICE_INLINE void loadInternalTwiddles64(S* data, bool stride)
  {
    UNROLL
    for (int i = 0; i < 7; i++) {
      WI[i] = data[((stride ? (threadIdx.x >> 3) : (threadIdx.x)) & 0x7) * (i + 1)];
    }
  }

  DEVICE_INLINE void loadInternalTwiddles32(S* data, bool stride)
  {
    UNROLL
    for (int i = 0; i < 7; i++) {
      WI[i] = data[2 * ((stride ? (threadIdx.x >> 4) : (threadIdx.x)) & 0x3) * (i + 1)];
    }
  }

  DEVICE_INLINE void loadInternalTwiddles16(S* data, bool stride)
  {
    UNROLL
    for (int i = 0; i < 7; i++) {
      WI[i] = data[4 * ((stride ? (threadIdx.x >> 5) : (threadIdx.x)) & 0x1) * (i + 1)];
    }
  }

  DEVICE_INLINE void loadInternalTwiddlesGeneric64(S* data, bool stride, bool inv)
  {
    UNROLL
    for (int i = 0; i < 7; i++) {
      uint32_t exp = ((stride ? (threadIdx.x >> 3) : (threadIdx.x)) & 0x7) * (i + 1);
      WI[i] = data[(inv && exp) ? 64 - exp : exp]; // if exp = 0 we also take exp and not 64-exp
    }
  }

  DEVICE_INLINE void loadInternalTwiddlesGeneric32(S* data, bool stride, bool inv)
  {
    UNROLL
    for (int i = 0; i < 7; i++) {
      uint32_t exp = 2 * ((stride ? (threadIdx.x >> 4) : (threadIdx.x)) & 0x3) * (i + 1);
      WI[i] = data[(inv && exp) ? 64 - exp : exp];
    }
  }

  DEVICE_INLINE void loadInternalTwiddlesGeneric16(S* data, bool stride, bool inv)
  {
    UNROLL
    for (int i = 0; i < 7; i++) {
      uint32_t exp = 4 * ((stride ? (threadIdx.x >> 5) : (threadIdx.x)) & 0x1) * (i + 1);
      WI[i] = data[(inv && exp) ? 64 - exp : exp];
    }
  }

  DEVICE_INLINE void loadExternalTwiddles64(S* data, uint32_t tw_order, uint32_t tw_log_order, stage_metadata s_meta)
  {
    data += tw_order * s_meta.ntt_inp_id + (s_meta.ntt_block_id & (tw_order - 1));

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      WE[i] = data[8 * i * tw_order + (1 << tw_log_order + 6) - 1];
    }
  }

  DEVICE_INLINE void loadExternalTwiddles32(S* data, uint32_t tw_order, uint32_t tw_log_order, stage_metadata s_meta)
  {
    data += tw_order * s_meta.ntt_inp_id * 2 + (s_meta.ntt_block_id & (tw_order - 1));

    UNROLL
    for (uint32_t j = 0; j < 2; j++) {
      UNROLL
      for (uint32_t i = 0; i < 4; i++) {
        WE[4 * j + i] = data[(8 * i + j) * tw_order + (1 << tw_log_order + 5) - 1];
      }
    }
  }

  DEVICE_INLINE void loadExternalTwiddles16(S* data, uint32_t tw_order, uint32_t tw_log_order, stage_metadata s_meta)
  {
    data += tw_order * s_meta.ntt_inp_id * 4 + (s_meta.ntt_block_id & (tw_order - 1));

    UNROLL
    for (uint32_t j = 0; j < 4; j++) {
      UNROLL
      for (uint32_t i = 0; i < 2; i++) {
        WE[2 * j + i] = data[(8 * i + j) * tw_order + (1 << tw_log_order + 4) - 1];
      }
    }
  }

  DEVICE_INLINE void loadExternalTwiddlesGeneric64(
    S* data, uint32_t tw_order, uint32_t tw_log_order, stage_metadata s_meta, uint32_t tw_log_size, bool inv)
  {
    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      uint32_t exp = (s_meta.ntt_inp_id + 8 * i) * (s_meta.ntt_block_id & (tw_order - 1))
                     << (tw_log_size - tw_log_order - 6);
      WE[i] = data[(inv && exp) ? ((1 << tw_log_size) - exp) : exp];
    }
  }

  DEVICE_INLINE void loadExternalTwiddlesGeneric32(
    S* data, uint32_t tw_order, uint32_t tw_log_order, stage_metadata s_meta, uint32_t tw_log_size, bool inv)
  {
    UNROLL
    for (uint32_t j = 0; j < 2; j++) {
      UNROLL
      for (uint32_t i = 0; i < 4; i++) {
        uint32_t exp = (s_meta.ntt_inp_id * 2 + 8 * i + j) * (s_meta.ntt_block_id & (tw_order - 1))
                       << (tw_log_size - tw_log_order - 5);
        WE[4 * j + i] = data[(inv && exp) ? ((1 << tw_log_size) - exp) : exp];
      }
    }
  }

  DEVICE_INLINE void loadExternalTwiddlesGeneric16(
    S* data, uint32_t tw_order, uint32_t tw_log_order, stage_metadata s_meta, uint32_t tw_log_size, bool inv)
  {
    UNROLL
    for (uint32_t j = 0; j < 4; j++) {
      UNROLL
      for (uint32_t i = 0; i < 2; i++) {
        uint32_t exp = (s_meta.ntt_inp_id * 4 + 8 * i + j) * (s_meta.ntt_block_id & (tw_order - 1))
                       << (tw_log_size - tw_log_order - 4);
        WE[2 * j + i] = data[(inv && exp) ? ((1 << tw_log_size) - exp) : exp];
      }
    }
  }

  DEVICE_INLINE void
  loadGlobalData(const E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id;
    }

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      X[i] = data[s_meta.th_stride * i * data_stride_u64];
    }
  }

  DEVICE_INLINE void loadGlobalDataColumnBatch(
    const E* data, uint32_t data_stride, uint32_t log_data_stride, stage_metadata s_meta, uint32_t batch_size)
  {
    const uint64_t data_stride_u64 = data_stride;
    data += ((s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id +
             (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size) *
              batch_size +
            s_meta.batch_id;

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      X[i] = data[s_meta.th_stride * i * data_stride_u64 * batch_size];
    }
  }

  DEVICE_INLINE void
  storeGlobalData(E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id;
    }

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      data[s_meta.th_stride * i * data_stride_u64] = X[i];
    }
  }

  DEVICE_INLINE void storeGlobalDataColumnBatch(
    E* data, uint32_t data_stride, uint32_t log_data_stride, stage_metadata s_meta, uint32_t batch_size)
  {
    const uint64_t data_stride_u64 = data_stride;
    data += ((s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id +
             (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size) *
              batch_size +
            s_meta.batch_id;

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      data[s_meta.th_stride * i * data_stride_u64 * batch_size] = X[i];
    }
  }

  DEVICE_INLINE void
  loadGlobalData32(const E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id * 2 +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id * 2;
    }

    UNROLL
    for (uint32_t j = 0; j < 2; j++) {
      UNROLL
      for (uint32_t i = 0; i < 4; i++) {
        X[4 * j + i] = data[(8 * i + j) * data_stride_u64];
      }
    }
  }

  DEVICE_INLINE void loadGlobalData32ColumnBatch(
    const E* data, uint32_t data_stride, uint32_t log_data_stride, stage_metadata s_meta, uint32_t batch_size)
  {
    const uint64_t data_stride_u64 = data_stride;
    data += ((s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id * 2 +
             (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size) *
              batch_size +
            s_meta.batch_id;

    UNROLL
    for (uint32_t j = 0; j < 2; j++) {
      UNROLL
      for (uint32_t i = 0; i < 4; i++) {
        X[4 * j + i] = data[(8 * i + j) * data_stride_u64 * batch_size];
      }
    }
  }

  DEVICE_INLINE void
  storeGlobalData32(E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id * 2 +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id * 2;
    }

    UNROLL
    for (uint32_t j = 0; j < 2; j++) {
      UNROLL
      for (uint32_t i = 0; i < 4; i++) {
        data[(8 * i + j) * data_stride_u64] = X[4 * j + i];
      }
    }
  }

  DEVICE_INLINE void storeGlobalData32ColumnBatch(
    E* data, uint32_t data_stride, uint32_t log_data_stride, stage_metadata s_meta, uint32_t batch_size)
  {
    const uint64_t data_stride_u64 = data_stride;
    data += ((s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id * 2 +
             (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size) *
              batch_size +
            s_meta.batch_id;

    UNROLL
    for (uint32_t j = 0; j < 2; j++) {
      UNROLL
      for (uint32_t i = 0; i < 4; i++) {
        data[(8 * i + j) * data_stride_u64 * batch_size] = X[4 * j + i];
      }
    }
  }

  DEVICE_INLINE void
  loadGlobalData16(const E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id * 4 +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id * 4;
    }

    UNROLL
    for (uint32_t j = 0; j < 4; j++) {
      UNROLL
      for (uint32_t i = 0; i < 2; i++) {
        X[2 * j + i] = data[(8 * i + j) * data_stride_u64];
      }
    }
  }

  DEVICE_INLINE void loadGlobalData16ColumnBatch(
    const E* data, uint32_t data_stride, uint32_t log_data_stride, stage_metadata s_meta, uint32_t batch_size)
  {
    const uint64_t data_stride_u64 = data_stride;
    data += ((s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id * 4 +
             (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size) *
              batch_size +
            s_meta.batch_id;

    UNROLL
    for (uint32_t j = 0; j < 4; j++) {
      UNROLL
      for (uint32_t i = 0; i < 2; i++) {
        X[2 * j + i] = data[(8 * i + j) * data_stride_u64 * batch_size];
      }
    }
  }

  DEVICE_INLINE void
  storeGlobalData16(E* data, uint32_t data_stride, uint32_t log_data_stride, bool strided, stage_metadata s_meta)
  {
    const uint64_t data_stride_u64 = data_stride;
    if (strided) {
      data += (s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id * 4 +
              (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size;
    } else {
      data += (uint64_t)s_meta.ntt_block_id * s_meta.ntt_block_size + s_meta.ntt_inp_id * 4;
    }

    UNROLL
    for (uint32_t j = 0; j < 4; j++) {
      UNROLL
      for (uint32_t i = 0; i < 2; i++) {
        data[(8 * i + j) * data_stride_u64] = X[2 * j + i];
      }
    }
  }

  DEVICE_INLINE void storeGlobalData16ColumnBatch(
    E* data, uint32_t data_stride, uint32_t log_data_stride, stage_metadata s_meta, uint32_t batch_size)
  {
    const uint64_t data_stride_u64 = data_stride;
    data += ((s_meta.ntt_block_id & (data_stride - 1)) + data_stride_u64 * s_meta.ntt_inp_id * 4 +
             (s_meta.ntt_block_id >> log_data_stride) * data_stride_u64 * s_meta.ntt_block_size) *
              batch_size +
            s_meta.batch_id;

    UNROLL
    for (uint32_t j = 0; j < 4; j++) {
      UNROLL
      for (uint32_t i = 0; i < 2; i++) {
        data[(8 * i + j) * data_stride_u64 * batch_size] = X[2 * j + i];
      }
    }
  }

  DEVICE_INLINE void ntt4_2()
  {
    UNROLL
    for (int i = 0; i < 2; i++) {
      ntt4(X[4 * i], X[4 * i + 1], X[4 * i + 2], X[4 * i + 3]);
    }
  }

  DEVICE_INLINE void ntt2_4()
  {
    UNROLL
    for (int i = 0; i < 4; i++) {
      ntt2(X[2 * i], X[2 * i + 1]);
    }
  }

  DEVICE_INLINE void ntt2(E& X0, E& X1)
  {
    E T;

    T = X0 + X1;
    X1 = X0 - X1;
    X0 = T;
  }

  DEVICE_INLINE void ntt4(E& X0, E& X1, E& X2, E& X3)
  {
    E T;

    T = X0 + X2;
    X2 = X0 - X2;
    X0 = X1 + X3;
    X1 = X1 - X3; // T has X0, X0 has X1, X2 has X2, X1 has X3

    X1 = X1 * WB[0];

    X3 = X2 - X1; // X'3 = (X0 - X2) - (X1 - X3) * WB[0]
    X1 = X2 + X1; // X'1 = (X0 - X2) + (X1 - X3) * WB[0]
    X2 = T - X0;  // X'2 = (X0 + X2) - (X1 + X3)
    X0 = T + X0;  // X'0 = (X0 + X2) + (X1 + X3)
  }

  // rbo version
  DEVICE_INLINE void ntt4rbo(E& X0, E& X1, E& X2, E& X3)
  {
    E T;

    T = X0 - X1;
    X0 = X0 + X1;
    X1 = X2 + X3;
    X3 = X2 - X3; // T has X0, X0 has X1, X2 has X2, X1 has X3

    X3 = X3 * WB[0];

    X2 = X0 - X1;
    X0 = X0 + X1;
    X1 = T + X3;
    X3 = T - X3;
  }

  DEVICE_INLINE void ntt8(E& X0, E& X1, E& X2, E& X3, E& X4, E& X5, E& X6, E& X7)
  {
    E T;

    // out of 56,623,104 possible mappings, we have:
    T = X3 - X7;
    X7 = X3 + X7;
    X3 = X1 - X5;
    X5 = X1 + X5;
    X1 = X2 + X6;
    X2 = X2 - X6;
    X6 = X0 + X4;
    X0 = X0 - X4;

    T = T * WB[1];
    X2 = X2 * WB[1];

    X4 = X6 + X1;
    X6 = X6 - X1;
    X1 = X3 + T;
    X3 = X3 - T;
    T = X5 + X7;
    X5 = X5 - X7;
    X7 = X0 + X2;
    X0 = X0 - X2;

    X1 = X1 * WB[0];
    X5 = X5 * WB[1];
    X3 = X3 * WB[2];

    X2 = X6 + X5;
    X6 = X6 - X5;
    X5 = X7 - X1;
    X1 = X7 + X1;
    X7 = X0 - X3;
    X3 = X0 + X3;
    X0 = X4 + T;
    X4 = X4 - T;
  }

  DEVICE_INLINE void ntt8win()
  {
    E T;

    T = X[3] - X[7];
    X[7] = X[3] + X[7];
    X[3] = X[1] - X[5];
    X[5] = X[1] + X[5];
    X[1] = X[2] + X[6];
    X[2] = X[2] - X[6];
    X[6] = X[0] + X[4];
    X[0] = X[0] - X[4];

    X[2] = X[2] * WB[0];

    X[4] = X[6] + X[1];
    X[6] = X[6] - X[1];
    X[1] = X[3] + T;
    X[3] = X[3] - T;
    T = X[5] + X[7];
    X[5] = X[5] - X[7];
    X[7] = X[0] + X[2];
    X[0] = X[0] - X[2];

    X[1] = X[1] * WB[1];
    X[5] = X[5] * WB[0];
    X[3] = X[3] * WB[2];

    X[2] = X[6] + X[5];
    X[6] = X[6] - X[5];

    X[5] = X[1] + X[3];
    X[3] = X[1] - X[3];

    X[1] = X[7] + X[5];
    X[5] = X[7] - X[5];
    X[7] = X[0] - X[3];
    X[3] = X[0] + X[3];
    X[0] = X[4] + T;
    X[4] = X[4] - T;
  }

  DEVICE_INLINE void SharedData64Columns8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x7 : threadIdx.x >> 3;
    uint32_t column_id = stride ? threadIdx.x >> 3 : threadIdx.x & 0x7;

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 64 + i * 8 + column_id] = X[i];
      } else {
        X[i] = shmem[ntt_id * 64 + i * 8 + column_id];
      }
    }
  }

  DEVICE_INLINE void SharedData64Rows8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x7 : threadIdx.x >> 3;
    uint32_t row_id = stride ? threadIdx.x >> 3 : threadIdx.x & 0x7;

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 64 + row_id * 8 + i] = X[i];
      } else {
        X[i] = shmem[ntt_id * 64 + row_id * 8 + i];
      }
    }
  }

  DEVICE_INLINE void SharedData32Columns8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0xf : threadIdx.x >> 2;
    uint32_t column_id = stride ? threadIdx.x >> 4 : threadIdx.x & 0x3;

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 32 + i * 4 + column_id] = X[i];
      } else {
        X[i] = shmem[ntt_id * 32 + i * 4 + column_id];
      }
    }
  }

  DEVICE_INLINE void SharedData32Rows8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0xf : threadIdx.x >> 2;
    uint32_t row_id = stride ? threadIdx.x >> 4 : threadIdx.x & 0x3;

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 32 + row_id * 8 + i] = X[i];
      } else {
        X[i] = shmem[ntt_id * 32 + row_id * 8 + i];
      }
    }
  }

  DEVICE_INLINE void SharedData32Columns4_2(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0xf : threadIdx.x >> 2;
    uint32_t column_id = (stride ? threadIdx.x >> 4 : threadIdx.x & 0x3) * 2;

    UNROLL
    for (uint32_t j = 0; j < 2; j++) {
      UNROLL
      for (uint32_t i = 0; i < 4; i++) {
        if (store) {
          shmem[ntt_id * 32 + i * 8 + column_id + j] = X[4 * j + i];
        } else {
          X[4 * j + i] = shmem[ntt_id * 32 + i * 8 + column_id + j];
        }
      }
    }
  }

  DEVICE_INLINE void SharedData32Rows4_2(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0xf : threadIdx.x >> 2;
    uint32_t row_id = (stride ? threadIdx.x >> 4 : threadIdx.x & 0x3) * 2;

    UNROLL
    for (uint32_t j = 0; j < 2; j++) {
      UNROLL
      for (uint32_t i = 0; i < 4; i++) {
        if (store) {
          shmem[ntt_id * 32 + row_id * 4 + 4 * j + i] = X[4 * j + i];
        } else {
          X[4 * j + i] = shmem[ntt_id * 32 + row_id * 4 + 4 * j + i];
        }
      }
    }
  }

  DEVICE_INLINE void SharedData16Columns8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x1f : threadIdx.x >> 1;
    uint32_t column_id = stride ? threadIdx.x >> 5 : threadIdx.x & 0x1;

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 16 + i * 2 + column_id] = X[i];
      } else {
        X[i] = shmem[ntt_id * 16 + i * 2 + column_id];
      }
    }
  }

  DEVICE_INLINE void SharedData16Rows8(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x1f : threadIdx.x >> 1;
    uint32_t row_id = stride ? threadIdx.x >> 5 : threadIdx.x & 0x1;

    UNROLL
    for (uint32_t i = 0; i < 8; i++) {
      if (store) {
        shmem[ntt_id * 16 + row_id * 8 + i] = X[i];
      } else {
        X[i] = shmem[ntt_id * 16 + row_id * 8 + i];
      }
    }
  }

  DEVICE_INLINE void SharedData16Columns2_4(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x1f : threadIdx.x >> 1;
    uint32_t column_id = (stride ? threadIdx.x >> 5 : threadIdx.x & 0x1) * 4;

    UNROLL
    for (uint32_t j = 0; j < 4; j++) {
      UNROLL
      for (uint32_t i = 0; i < 2; i++) {
        if (store) {
          shmem[ntt_id * 16 + i * 8 + column_id + j] = X[2 * j + i];
        } else {
          X[2 * j + i] = shmem[ntt_id * 16 + i * 8 + column_id + j];
        }
      }
    }
  }

  DEVICE_INLINE void SharedData16Rows2_4(E* shmem, bool store, bool high_bits, bool stride)
  {
    uint32_t ntt_id = stride ? threadIdx.x & 0x1f : threadIdx.x >> 1;
    uint32_t row_id = (stride ? threadIdx.x >> 5 : threadIdx.x & 0x1) * 4;

    UNROLL
    for (uint32_t j = 0; j < 4; j++) {
      UNROLL
      for (uint32_t i = 0; i < 2; i++) {
        if (store) {
          shmem[ntt_id * 16 + row_id * 2 + 2 * j + i] = X[2 * j + i];
        } else {
          X[2 * j + i] = shmem[ntt_id * 16 + row_id * 2 + 2 * j + i];
        }
      }
    }
  }

  DEVICE_INLINE void twiddlesInternal()
  {
    UNROLL
    for (int i = 1; i < 8; i++) {
      X[i] = X[i] * WI[i - 1];
    }
  }

  DEVICE_INLINE void twiddlesExternal()
  {
    UNROLL
    for (int i = 0; i < 8; i++) {
      X[i] = X[i] * WE[i];
    }
  }
};

#endif