#include "fields/field_config.cuh"
using namespace field_config;

#include "fri.cu"
#include "utils/utils.h"
#include "fields/point.cuh"

namespace fri {
  /**
   * Extern "C" version of [fold_line](@ref fold_line) function with the following values of
   * template parameters (where the field is given by `-DFIELD` env variable during build):
   *  - `E` is the extension field type used for evaluations and alpha
   *  - `S` is the scalar field type used for domain elements
   * @param line_eval Pointer to the array of evaluations on the line
   * @param domain_elements Pointer to the array of domain elements
   * @param alpha The folding factor
   * @param folded_evals Pointer to the array where folded evaluations will be stored
   * @param n The number of evaluations
   * @param ctx The device context; if the stream is not 0, then everything is run async
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(FIELD, fold_line)(
    q_extension_t* line_eval,
    scalar_t* domain_elements,
    q_extension_t alpha,
    q_extension_t* folded_evals,
    uint64_t n,
    FriConfig& cfg)
  {
    return fri::fold_line(line_eval, domain_elements, alpha, folded_evals, n, cfg);
  };

  extern "C" hipError_t CONCAT_EXPAND(FIELD, fold_line_new)(
    q_extension_t* line_eval,
    uint64_t line_domain_initial_index,
    uint32_t line_domain_log_size,
    q_extension_t alpha,
    q_extension_t* folded_evals,
    uint64_t n,
    FriConfig& cfg)
  {
    circle_math::LineDomain<fp_config, scalar_t> line_domain = circle_math::LineDomain<fp_config, scalar_t>(line_domain_initial_index, line_domain_log_size);
    scalar_t* domain_elements;
    hipMalloc(&domain_elements, line_domain.coset.size() * sizeof(scalar_t));
    line_domain.get_twiddles(domain_elements);
    cfg.are_domain_elements_on_device = true;
    return fri::fold_line(line_eval, domain_elements, alpha, folded_evals, n, cfg);
  };

  /**
   * Extern "C" version of [fold_circle_into_line](@ref fold_circle_into_line) function with the following values of
   * template parameters (where the field is given by `-DFIELD` env variable during build):
   *  - `E` is the extension field type used for evaluations and alpha
   *  - `S` is the scalar field type used for domain elements
   * @param circle_evals Pointer to the array of evaluations on the circle
   * @param domain_elements Pointer to the array of domain elements
   * @param alpha The folding factor
   * @param folded_line_evals Pointer to the array where folded evaluations will be stored
   * @param n The number of evaluations
   * @param ctx The device context; if the stream is not 0, then everything is run async
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(FIELD, fold_circle_into_line)(
    q_extension_t* circle_evals,
    scalar_t* domain_elements,
    q_extension_t alpha,
    q_extension_t* folded_line_evals,
    uint64_t n,
    FriConfig& cfg)
  {
    return fri::fold_circle_into_line(circle_evals, domain_elements, alpha, folded_line_evals, n, cfg);
  };

  extern "C" hipError_t CONCAT_EXPAND(FIELD, fold_circle_into_line_new)(
    q_extension_t* circle_evals,
    uint64_t domain_initial_index,
    uint32_t domain_log_size,
    q_extension_t alpha,
    q_extension_t* folded_line_evals,
    uint64_t n,
    FriConfig& cfg)
  {
    domain_t domain(coset_t(domain_initial_index, domain_log_size));
    scalar_t* domain_elements;
    hipMalloc(&domain_elements, domain.coset.size() * sizeof(scalar_t));
    domain.get_twiddles(domain_elements);
    cfg.are_domain_elements_on_device = true;
    return fri::fold_circle_into_line(circle_evals, domain_elements, alpha, folded_line_evals, n, cfg);
  };
} // namespace fri
