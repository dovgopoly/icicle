#include "fields/field_config.cuh"
using namespace field_config;

#include "fri.cu"
#include "utils/utils.h"
#include "fields/point.cuh"

namespace fri {
  /**
   * Extern "C" version of [fold_line](@ref fold_line) function with the following values of
   * template parameters (where the field is given by `-DFIELD` env variable during build):
   *  - `E` is the extension field type used for evaluations and alpha
   *  - `S` is the scalar field type used for domain elements
   * @param line_eval Pointer to the array of evaluations on the line
   * @param domain_elements Pointer to the array of domain elements
   * @param alpha The folding factor
   * @param folded_evals Pointer to the array where folded evaluations will be stored
   * @param n The number of evaluations
   * @param ctx The device context; if the stream is not 0, then everything is run async
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(FIELD, fold_line)(
    q_extension_t* line_eval,
    scalar_t* domain_elements,
    q_extension_t alpha,
    q_extension_t* folded_evals,
    uint64_t n,
    FriConfig& cfg)
  {
    return fri::fold_line(line_eval, domain_elements, alpha, folded_evals, n, cfg);
  };

  extern "C" hipError_t CONCAT_EXPAND(FIELD, fold_line_new)(
    q_extension_t* line_eval,
    uint64_t line_domain_initial_index,
    uint32_t line_domain_log_size,
    q_extension_t alpha,
    q_extension_t* folded_evals,
    uint64_t n,
    FriConfig& cfg)
  {
    line_t line_domain(line_domain_initial_index, line_domain_log_size);
    line_t test_domain(coset_t::half_odds(line_domain_log_size));
    scalar_t* domain_elements;
    line_domain.get_twiddles(&domain_elements);
    cfg.are_domain_elements_on_device = true;
    return fri::fold_line(line_eval, domain_elements, alpha, folded_evals, n, cfg);
  };

  /**
   * Extern "C" version of [fold_circle_into_line](@ref fold_circle_into_line) function with the following values of
   * template parameters (where the field is given by `-DFIELD` env variable during build):
   *  - `E` is the extension field type used for evaluations and alpha
   *  - `S` is the scalar field type used for domain elements
   * @param circle_evals Pointer to the array of evaluations on the circle
   * @param domain_elements Pointer to the array of domain elements
   * @param alpha The folding factor
   * @param folded_line_evals Pointer to the array where folded evaluations will be stored
   * @param n The number of evaluations
   * @param ctx The device context; if the stream is not 0, then everything is run async
   * @return `hipSuccess` if the execution was successful and an error code otherwise.
   */
  extern "C" hipError_t CONCAT_EXPAND(FIELD, fold_circle_into_line)(
    q_extension_t* circle_evals,
    scalar_t* domain_elements,
    q_extension_t alpha,
    q_extension_t* folded_line_evals,
    uint64_t n,
    FriConfig& cfg)
  {
    return fri::fold_circle_into_line(circle_evals, domain_elements, alpha, folded_line_evals, n, cfg);
  };

  extern "C" hipError_t CONCAT_EXPAND(FIELD, fold_circle_into_line_new)(
    q_extension_t* circle_evals,
    uint64_t domain_initial_index,
    uint32_t domain_log_size,
    q_extension_t alpha,
    q_extension_t* folded_line_evals,
    uint64_t n,
    FriConfig& cfg)
  {
    domain_t test_domain(domain_log_size + 1);
    domain_t domain(coset_t(domain_initial_index, domain_log_size));
    scalar_t* domain_elements;
    domain.get_twiddles(&domain_elements);
    cfg.are_domain_elements_on_device = true;
    return fri::fold_circle_into_line(circle_evals, domain_elements, alpha, folded_line_evals, n, cfg);
  };

  extern "C" hipError_t CONCAT_EXPAND(FIELD, precompute_fri_twiddles)(
    uint32_t log_size
    )
  {
    CHK_INIT_IF_RETURN();
    for(uint32_t i = 2; i <= log_size; ++i) {
      coset_t coset = coset_t::half_odds(i);
      domain_t domain(coset);
      domain.compute_twiddles();
      line_t line_domain(coset);
      line_domain.compute_twiddles();
    }
    return CHK_LAST();
  };
} // namespace fri
